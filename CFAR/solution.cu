#include <hip/hip_runtime.h>
#include<stdlib.h>
#include <wb.h>
#include <iostream>

#include "kernel.cu"


int main() 
{
	int debug = 0;
	int version = 0;
	const int numCells = 256 * 1024 * 20; // Number of cells in the range profile

	hipEvent_t astartEvent, astopEvent;
	float aelapsedTime;
	hipEventCreate(&astartEvent);
	hipEventCreate(&astopEvent);

    // Generate synthetic input data for testing

    float* inputData = new float[numCells];

	for (int i = 0; i < numCells; ++i)
	{
		inputData[i] = 20 + std::rand() % 20;
	}
	inputData[1152] = 90;
	inputData[1153] = 100;
	inputData[1154] = 90;

	if (debug == 1) {
	for (int j = 0; j < 10; ++j)
	{
		printf("inputdata[%d]: %f \n", j, inputData[j]);
	} }

    // Allocate memory on the GPU
    float* d_inputData;
    int* d_detections;
	int* d_sharedDetections;
    hipMalloc(&d_inputData, numCells * sizeof(float));
    hipMalloc(&d_detections, numCells * sizeof(int));
    hipMalloc(&d_sharedDetections, numCells * sizeof(int));

    // Copy input data to the GPU
    hipMemcpy(d_inputData, inputData, numCells * sizeof(float), hipMemcpyHostToDevice);

    // Launch the CFAR kernel
    const unsigned int blockSize = 256;
    const unsigned int numBlocks = (numCells + blockSize - 1) / blockSize;

	//*****************Naive Version***********************
	if (version == 0) {
  	hipEventRecord(astartEvent, 0);
    cfarKernel<<<numBlocks, blockSize>>>(d_inputData, d_detections, numCells);

	hipEventRecord(astopEvent, 0);
	hipEventSynchronize(astopEvent);
	hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
	printf("\n");
	printf("Naive Version total compute time (ms) %f ",aelapsedTime);
	printf("\n"); }

	//*****************Shared Memory Version***************
	if (version == 0) {
  	hipEventRecord(astartEvent, 0);
    cfarKernelShared<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(d_inputData, d_sharedDetections, numCells);

	hipEventRecord(astopEvent, 0);
	hipEventSynchronize(astopEvent);
	hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
	printf("\n");
	printf("Shared Version total compute time (ms) %f ",aelapsedTime);
	printf("\n"); }

    // Copy results back to the CPU
    int* detections = new int[numCells];
    int* sharedDetections = new int[numCells];
    hipMemcpy(detections, d_detections, numCells * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(sharedDetections, d_sharedDetections, numCells * sizeof(int), hipMemcpyDeviceToHost);

	if (debug == 1) {
	printf("\n");
	for (int j = 0; j < 50; ++j)
	{
		printf("detections[%d]: %d \n", j, sharedDetections[j]);
	}
	printf("\n"); }
	if (debug == 1) 
	{
		int numDetect = 0;
		printf("\n");
		for (int i = 0; i < numCells; ++i)
		{
			if (sharedDetections[i] == 1) 
			{
				numDetect++;
				//printf("detection at: %d \n", i); 
			}

		}
		printf("Number of detections: %d \n", numDetect);
	}


    // Clean up
	hipEventDestroy(astopEvent);
	hipEventDestroy(astartEvent);
    delete[] inputData;
    delete[] detections;
    hipFree(d_inputData);
    hipFree(d_detections);
    hipFree(d_sharedDetections);


    return 0;
}

#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <sstream>
#include <hip/hip_runtime.h>

__global__ void kalmanFilter(float *stateEstimates, float *covariances, const float *measurements, float processNoise, float measurementNoise, int numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements) {
        // Predict
        float predictedState = stateEstimates[idx];
        float predictedCovariance = covariances[idx] + processNoise;

        // Update
        float innovation = measurements[idx] - predictedState;
        float innovationCovariance = predictedCovariance + measurementNoise;
        float kalmanGain = predictedCovariance / innovationCovariance;

        stateEstimates[idx] = predictedState + kalmanGain * innovation;
        covariances[idx] = (1 - kalmanGain) * predictedCovariance;
    }
}

void checkCudaError(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        std::cerr << "CUDA Error: " << message << " - " << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }
}


int main() {
    std::string line;
    std::ifstream file("./radarDataMatlab.csv");
    std::vector<float> measurements;

    if (file.is_open()) {
        while (getline(file, line)) {
            std::stringstream linestream(line);
            std::string cell;
            getline(linestream, cell, ',');  // Skip time column
            getline(linestream, cell, ',');
            measurements.push_back(std::stof(cell));
        }
        file.close();
    } else {
        std::cerr << "Unable to open file" << std::endl;
        return EXIT_FAILURE;
    }

    int n = measurements.size();
    std::vector<float> stateEstimate(n, 0);  // Initial state estimates
    std::vector<float> covarianceEstimate(n, 1);  // Initial covariance estimates

    float *d_stateEstimate, *d_covarianceEstimate, *d_measurements;
    checkCudaError(hipMalloc(&d_stateEstimate, n * sizeof(float)), "allocating d_stateEstimate");
    checkCudaError(hipMalloc(&d_covarianceEstimate, n * sizeof(float)), "allocating d_covarianceEstimate");
    checkCudaError(hipMalloc(&d_measurements, n * sizeof(float)), "allocating d_measurements");

    checkCudaError(hipMemcpy(d_stateEstimate, stateEstimate.data(), n * sizeof(float), hipMemcpyHostToDevice), "copying to d_stateEstimate");
    checkCudaError(hipMemcpy(d_covarianceEstimate, covarianceEstimate.data(), n * sizeof(float), hipMemcpyHostToDevice), "copying to d_covarianceEstimate");
    checkCudaError(hipMemcpy(d_measurements, measurements.data(), n * sizeof(float), hipMemcpyHostToDevice), "copying to d_measurements");

    float processNoise = 1e-5;  // Example process noise
    float measurementNoise = 1e-5;  // Example measurement noise

    // Set up CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start recording
    hipEventRecord(start);

    // Launch the kernel
    int threadsPerBlock = 8;
    int blocksPerGrid = 512;
    kalmanFilter<<<blocksPerGrid, threadsPerBlock>>>(d_stateEstimate, d_covarianceEstimate, d_measurements, processNoise, measurementNoise, n);

    // Stop recording
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate and print the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " milliseconds\n and the number of blocks is " << blocksPerGrid << " and threads is" << threadsPerBlock << "\n";

    checkCudaError(hipMemcpy(stateEstimate.data(), d_stateEstimate, n * sizeof(float), hipMemcpyDeviceToHost), "copying from d_stateEstimate");
    checkCudaError(hipMemcpy(covarianceEstimate.data(), d_covarianceEstimate, n * sizeof(float), hipMemcpyDeviceToHost), "copying from d_covarianceEstimate");

    hipFree(d_stateEstimate);
    hipFree(d_covarianceEstimate);
    hipFree(d_measurements);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}